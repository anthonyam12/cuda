
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

int main()
{
	const int kb = 1024;
	const int mb = kb*kb;
	cout << "NBody.GPU" << endl << "=============" << endl << endl;
	//cout < "CUDA Version:\tv" << CUDART_VERSION << endl;
	

	int devCount;
	hipGetDeviceCount(&devCount);
	cout << "CUDA Devices: " << endl << endl;
	for (int i = 0; i < devCount; i++) 
	{
		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, i);
		cout << i << ": " << props.name << ": " << props.major << "." << props.minor << endl;
        cout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << endl;
        cout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << endl;
        cout << "  Constant memory: " << props.totalConstMem / kb << "kb" << endl;
        cout << "  Block registers: " << props.regsPerBlock << endl << endl;

        cout << "  Warp size:         " << props.warpSize << endl;
        cout << "  Threads per block: " << props.maxThreadsPerBlock << endl;
        cout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1]  << ", " << props.maxThreadsDim[2] << " ]" << endl;
        cout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1]  << ", " << props.maxGridSize[2] << " ]" << endl;
	}

	return 0;
}
